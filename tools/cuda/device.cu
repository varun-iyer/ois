#include "hip/hip_runtime.h"
#include "device.h"

__global__ void par_reduce(size_t n, double *d, double *c1, double *c2) {
	uint tid = threadIdx.x;
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	sm[tid] = i<n ? c1[i] * c2[i] : 0; // copy to SM
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s > 0; s>>=1) {
		if(tid < s) {
			sm[tid] += sm[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0) atomicAdd(d, sm[0]);
	// d[blockIdx.x] is the sum of the block
}

__global__ void par_reduce_mask(size_t n, double *d, double *c1, double *c2, char *mask) {
	uint tid = threadIdx.x;
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	sm[tid] = (mask[i] || i >= n) ? 0 : c1[i] * c2[i]; // copy to SM
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s > 0; s>>=1) {
		if(tid < s) {
			sm[tid] += sm[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0) atomicAdd(d, sm[0]);
}
