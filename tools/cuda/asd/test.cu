#include "hip/hip_runtime.h"
#include "util.h"

__global__ void my_dbl_sq(float *val){
  *val = 2*my_square(*val);
}

float dbl_sq(float val){
  float *d_val, h_val;
  hipMalloc(&d_val, sizeof(float));
  h_val = val;
  hipMemcpy(d_val, &h_val, sizeof(float), hipMemcpyHostToDevice);
  my_dbl_sq<<<1,1>>>(d_val);
  hipMemcpy(&h_val, d_val, sizeof(float), hipMemcpyDeviceToHost);
  return h_val;
}
