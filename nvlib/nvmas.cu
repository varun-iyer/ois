#include <stdio.h>
#include "device.h"
#include "nvmas.h"

#define MAX_THREADS 1024
#define SMSIZE (sizeof(double) * MAX_THREADS)

extern "C" double multiply_and_sum(size_t nsize, double* h_C1, double* h_C2) {
	double *d_C1, *d_C2, *d;

	hipMalloc(&d_C1, nsize * sizeof(double));
	hipMalloc(&d_C2, nsize * sizeof(double));
	hipMalloc(&d, nsize * sizeof(double));

	hipMemcpy(d_C1, h_C1, nsize * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_C2, h_C2, nsize * sizeof(double), hipMemcpyHostToDevice);

	dim3 dimBlock(MAX_THREADS, 1, 1);
	dim3 dimGrid(nsize / MAX_THREADS + 1, 1, 1);
	par_reduce<<<dimGrid, dimBlock, SMSIZE>>>(nsize, d, d_C1, d_C2);

	double h_r;
	hipMemcpy(&h_r, d, sizeof(double), hipMemcpyDeviceToHost);
	hipFree(d_C1);
	hipFree(d_C2);
	hipFree(d);

    return h_r;
}

extern "C" double multiply_and_sum_mask(size_t nsize, double* h_C1, double* h_C2, char *h_m) {
	double *d_C1, *d_C2, *d;
	char *d_m;

	hipMalloc(&d_C1, nsize * sizeof(double));
	hipMalloc(&d_C2, nsize * sizeof(double));
	hipMalloc(&d_m, nsize * sizeof(char));
	hipMalloc(&d, nsize * sizeof(double));

	hipMemcpy(d_C1, h_C1, nsize * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_C2, h_C2, nsize * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_m, h_m, nsize * sizeof(char), hipMemcpyHostToDevice);

	dim3 dimBlock(MAX_THREADS, 1, 1);
	dim3 dimGrid(nsize / MAX_THREADS + 1, 1, 1);
	par_reduce_mask<<<dimGrid, dimBlock, SMSIZE>>>(nsize, d, d_C1, d_C2, d_m);

	double h_r;
	hipMemcpy(&h_r, d, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_C1);
	hipFree(d_C2);
	hipFree(d);
    return h_r;
}
